#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "../include/CUDA_wrappers.hpp"
#include <host_defines.h>
#include <>

// функция вычисления точки
__global__ void compute_point(double* x_down, double* x_up, double* y_down, double* y_up, unsigned int* iteration_count, unsigned int* width, unsigned int* height, unsigned int* matrix) {
    double re = *x_down + (*x_up - *x_down) * ((blockIdx.x % *width) + 0.5) / (double) *width;
    double im = *y_up - (*y_up - *y_down) * ((blockIdx.x / *width) + 0.5) / (double) *height;
    double re_curr = 0;
    double im_curr = 0;
    // проверка на принадлежность главной картиоиде

    // проверка точки
    for(unsigned int iteration = 1; iteration <= *iteration_count; ++iteration) {
        re_curr = re_curr * re_curr - im_curr * im_curr + re;
        im_curr = 2 * re_curr * im_curr + im;
        if(re_curr * re_curr + im_curr * im_curr >= 4.0) {
            matrix[blockIdx.x] = iteration;
            break;
        }
    }
    matrix[blockIdx.x] = 0; // если точка все еще не вышла за границу
}

// функция, которая будет запускать вычисления
void compute_matrix(unsigned int* matrix, double x_down, double x_up, double y_down, double y_up, unsigned int width, unsigned int height, unsigned int iteration_count) {
    // просчитываем изображение на CUDA
    // выделяем память на видеокарте
    double* dev_x_down;
    double* dev_x_up;
    double* dev_y_down;
    double* dev_y_up;
    unsigned int* dev_iteration_count;
    unsigned int* dev_width;
    unsigned int* dev_height;
    unsigned int* dev_matrix;
    hipMalloc((void**)&dev_x_down, sizeof(double));
    hipMalloc((void**)&dev_x_up, sizeof(double));
    hipMalloc((void**)&dev_y_down, sizeof(double));
    hipMalloc((void**)&dev_y_up, sizeof(double));
    hipMalloc((void**)&dev_iteration_count, sizeof(unsigned int));
    hipMalloc((void**)&dev_width, sizeof(unsigned int));
    hipMalloc((void**)&dev_height, sizeof(unsigned int));
    hipMalloc((void**)&dev_matrix, width * height * sizeof(unsigned int));
    // переносим данные в память на видеокарте
    hipMemcpy(dev_x_down, &x_down, sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_x_up, &x_up, sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_y_down, &y_down, sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_y_up, &y_up, sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_iteration_count, &iteration_count, sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(dev_width, &width, sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(dev_height, &height, sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(dev_matrix, &matrix, width * height * sizeof(unsigned int), hipMemcpyHostToDevice);
    // запускаем ядро
    compute_point<<< (width * height), 1 >>>(dev_x_down, dev_x_up, dev_y_down, dev_y_up, dev_iteration_count, dev_width, dev_height, dev_matrix);
    // event
    hipEvent_t syncEvent;
    hipEventCreate(&syncEvent);    //Создаем event
    hipEventRecord(syncEvent, 0);  //Записываем event
    hipEventSynchronize(syncEvent);  //Синхронизируем event
    // забираем данные из памяти видеокарты
    hipMemcpy(matrix, dev_matrix, width * height, hipMemcpyDeviceToHost);
    // освобождаем память на видеокарте
    hipFree(dev_x_down);
    hipFree(dev_x_up);
    hipFree(dev_y_down);
    hipFree(dev_y_up);
    hipFree(dev_iteration_count);
    hipFree(dev_width);
    hipFree(dev_height);
    hipFree(dev_matrix);
}
