#include "hip/hip_runtime.h"
#include <host_defines.h>
#include <>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include "../include/CUDA_wrappers.hpp"

#define PRECISION double

// функция вычисления точки
__global__ void compute_point(PRECISION* x_down, PRECISION* x_up, PRECISION* y_down, PRECISION* y_up, unsigned int* iteration_count, unsigned int* width, unsigned int* height, unsigned int* matrix) {
    unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;
    PRECISION re = *x_down + (*x_up - *x_down) * (col + 0.5) / (PRECISION) (*width);
    PRECISION im = *y_up - (*y_up - *y_down) * (row + 0.5) / (PRECISION) (*height);
    PRECISION re_curr = 0.0;
    PRECISION im_curr = 0.0;
    // проверка на принадлежность главной картиоиде

    // проверка точки
    if(row >= *height || col >= *width) {
        return;
    }
    PRECISION re_temp;
    for(unsigned int iteration = 1; iteration <= *iteration_count; ++iteration) {
        re_temp = re_curr * re_curr - im_curr * im_curr + re;
        im_curr = 2.0 * re_curr * im_curr + im;
        re_curr = re_temp;
        if(re_curr * re_curr + im_curr * im_curr >= 4.0) {
            matrix[row * (*width) + col] = iteration;
            return;
        }
    }
    matrix[row * (*width) + col] = 0; // если точка все еще не вышла за границу
}

// функция, которая будет запускать вычисления
void compute_matrix(unsigned int* matrix, PRECISION x_down, PRECISION x_up, PRECISION y_down, PRECISION y_up, unsigned int width, unsigned int height, unsigned int iteration_count) {
    // просчитываем изображение на CUDA
    // выделяем память на видеокарте
    PRECISION* dev_x_down;
    PRECISION* dev_x_up;
    PRECISION* dev_y_down;
    PRECISION* dev_y_up;
    unsigned int* dev_width;
    unsigned int* dev_height;
    unsigned int* dev_matrix;
    unsigned int* dev_iteration_count;
    hipMalloc((void**)&dev_x_down, sizeof(PRECISION));
    hipMalloc((void**)&dev_x_up, sizeof(PRECISION));
    hipMalloc((void**)&dev_y_down, sizeof(PRECISION));
    hipMalloc((void**)&dev_y_up, sizeof(PRECISION));
    hipMalloc((void**)&dev_width, sizeof(unsigned int));
    hipMalloc((void**)&dev_height, sizeof(unsigned int));
    hipMalloc((void**)&dev_iteration_count, sizeof(unsigned int));
    hipMalloc((void**)&dev_matrix, width * height * sizeof(unsigned int));
    // переносим данные в память на видеокарте
    hipMemcpy(dev_x_down, &x_down, sizeof(PRECISION), hipMemcpyHostToDevice);
    hipMemcpy(dev_x_up, &x_up, sizeof(PRECISION), hipMemcpyHostToDevice);
    hipMemcpy(dev_y_down, &y_down, sizeof(PRECISION), hipMemcpyHostToDevice);
    hipMemcpy(dev_y_up, &y_up, sizeof(PRECISION), hipMemcpyHostToDevice);
    hipMemcpy(dev_width, &width, sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(dev_height, &height, sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(dev_iteration_count, &iteration_count, sizeof(unsigned int), hipMemcpyHostToDevice);
    // запускаем ядро
    dim3 block_size(16, 16);
    dim3 grid_size(ceil((double) width / (double) block_size.x), ceil((double) height / (double) block_size.y));
    compute_point<<<grid_size, block_size>>>(dev_x_down, dev_x_up, dev_y_down, dev_y_up, dev_iteration_count, dev_width, dev_height, dev_matrix);
    // забираем данные из памяти видеокарты
    hipMemcpy(matrix, dev_matrix, width * height * sizeof(unsigned int), hipMemcpyDeviceToHost);
    // освобождаем память на видеокарте
    hipFree(dev_x_down);
    hipFree(dev_x_up);
    hipFree(dev_y_down);
    hipFree(dev_y_up);
    hipFree(dev_width);
    hipFree(dev_height);
    hipFree(dev_iteration_count);
    hipFree(dev_matrix);
}
